#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>

/*		-- TRY CATCH PRINT EXCPEPTION pero CUDA --		*/
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t code, const char * file, int line, bool abort = true){
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*		-- FUNCIONES USADAS POR CPU --		*/

int randInRange(int lower, int upper) {
    return (rand() % (upper - lower + 1)) + lower;
}

void inicializar(short * matrix, int filas, int columnas) {
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j += 4) {

        	//estados
            int random = randInRange(0, 100);
            if (random <= 50) {
                matrix[i * columnas + j] = 4;
                matrix[i * columnas + j + 3] = -1;
            } else {
                random = randInRange(0, 100);
                if (random <= 20) {
                    matrix[i * columnas + j] = 2;
                    matrix[i * columnas + j + 3] = 6;
                } else if (random <= 70) {
                    matrix[i * columnas + j] = 3;
                    matrix[i * columnas + j + 3] = randInRange(0, 6);
                } else {
                    matrix[i * columnas + j] = 1;
                    matrix[i * columnas + j + 3] = randInRange(6, 8);
                }
            }

            //edades
            random = randInRange(0, 100);
            if (random <= 30) {
                matrix[i * columnas + j + 1] = randInRange(0, 104);		//104 semanas = 2 años
            } else if (random <= 84) {
                matrix[i * columnas + j + 1] = randInRange(105, 1976);	//1976 semanas = 38 años
            } else {
                matrix[i * columnas + j + 1] = randInRange(1977, 3640);	//3640 semanas = 70 años
            }

            //heridas
            matrix[i * columnas + j + 2] = randInRange(0, 1);
        }
    }
}

/*		-- FUNCIONES USADAS POR GPU --		*/

__device__ int randomDeviceGenerator(int seed, int lower, int upper) {
	//adaptacion simple de Multiply-with-carry pseudorandom number generator
	//https://en.wikipedia.org/wiki/Multiply-with-carry_pseudorandom_number_generator
    return ((seed * (threadIdx.x + 1) * (blockIdx.x + 1)) % (upper - lower + 1)) + lower;
}

__device__ short * vecindarioDeMoore(short * matrix, int filas, int columnas, int posX, int posY) {

    short * vecindario = (short * ) malloc(3 * 12 * sizeof(short));

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 9; j += 4) {
            int coordX = posX - 4 + j;
            int coordY = posY - 1 + i;
            if ((coordX >= 0) && (coordY >= 0)) {

            	/*
	            caso 1 = ambos positivos
	            caso 2 = alguno es negativo
	            caso 3 = alguno se paso de rango
	            se asigna -1 a todos los atributos de un vecino que no existe
	            */

                if ((coordX < columnas) && (coordY < filas)) {
                    //caso 1
                    vecindario[i * 12 + j] = matrix[coordY * columnas + coordX];
                    vecindario[i * 12 + j + 1] = matrix[coordY * columnas + coordX + 1];
                    vecindario[i * 12 + j + 2] = matrix[coordY * columnas + coordX + 2];
                    vecindario[i * 12 + j + 3] = matrix[coordY * columnas + coordX + 3];
                } else {
                    //caso 3
                    vecindario[i * 12 + j] = -1;
                    vecindario[i * 12 + j + 1] = -1;
                    vecindario[i * 12 + j + 2] = -1;
                    vecindario[i * 12 + j + 3] = -1;
                }
            } else {
                //caso 2
                vecindario[i * 12 + j] = -1;
                vecindario[i * 12 + j + 1] = -1;
                vecindario[i * 12 + j + 2] = -1;
                vecindario[i * 12 + j + 3] = -1;
            }
        }
    }
    return vecindario;
}

__device__ float susceptibilidad(int edad, int heridas) {
    float retorno;

    if (edad < 104) {			//104 semanas = 2 años
        retorno = 0.3;
    } else if (edad < 1976) {	//1976 semanas = 38 años
        retorno = 0.2;
    } else {
        retorno = 0.5;
    }
    if (heridas) {
        retorno += 0.15;
    }
    return retorno;
}

__device__ float porcentajeConSintomas(short * matrix) {
    int cont = 0;

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 12; j += 4) {
            if (((i != 1) && (j != 4)) && (matrix[i * 12 + j] == 2)) {
                cont++;
            }
        }
    }
    return cont / 8;
}

void printMatrix(short * matrix, int filas, int columnas) {
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j += 4) {
            printf(" -- %d %d %d %d", matrix[i * columnas + j], matrix[i * columnas + j + 1], matrix[i * columnas + j + 2], matrix[i * columnas + j + 3]);
        }
        printf("\n");
    }
    printf("\n");
}

__device__ void calcularEstado(int i, int j, short * matrix, short * matrixProxima, int filas, int semilla1, int semilla2, int semilla3, int semilla4){
	//printf("bloque %d thread %d está trabajando la fila %d columna %d\n", blockIdx.x, threadIdx.x, i,j);
	
	j *= 4; 						//una columna esta compuesta de 4 shorts
	int columnas = filas * 4;		//entonces existen (filas * 4) columnas
	int random;

	/*
	cada árbol esta representado por 4 posiciones consecutivas de memoria

	-- posición 0 => estado
	-- posición 1 => edad
	-- posición 2 => heridas (1 si / 0 no)
	-- posición 3 => contador auxiliar multiproposito
	*/

	/* -- INICIO DE: CALCULAR EL PROXIMO ESTADO -- */

    matrixProxima[i * columnas + j + 1] = matrix[i * columnas + j + 1] + 1; //edad++
    switch (matrix[i * columnas + j]) {
        case 0: //podado
            if (matrix[i * columnas + j + 3] == 7) {
                matrixProxima[i * columnas + j] = 4;
                matrixProxima[i * columnas + j + 3] = -1;
            } else {
                matrixProxima[i * columnas + j + 3] = matrix[i * columnas + j + 3] + 1;
            }
        break;

        case 1:
            random = randomDeviceGenerator(semilla1, 0, 100);
            if (matrix[i * columnas + j + 1] <= 104) {			//104 semanas = 2 años
                if (random > 1) {
                    matrixProxima[i * columnas + j] = 4;
                    matrixProxima[i * columnas + j + 3] = -1;
                } else {
                    matrixProxima[i * columnas + j] = 0;
                    matrixProxima[i * columnas + j + 3] = -1;
                }
            } else if (matrix[i * columnas + j + 1] <= 1976) {	//1976 semanas = 38 años
                if (random > 10) {
                    matrixProxima[i * columnas + j] = 4;
                    matrixProxima[i * columnas + j + 3] = -1;
                } else {
                    matrixProxima[i * columnas + j] = 0;
                    matrixProxima[i * columnas + j + 3] = -1;
                }
            } else {
                if (random > 45) {
                    matrixProxima[i * columnas + j] = 4;
                    matrixProxima[i * columnas + j + 3] = -1;
                } else {
                    matrixProxima[i * columnas + j] = 4;
                    matrixProxima[i * columnas + j + 1] = 52;
                    matrixProxima[i * columnas + j + 3] = -1;
                }
            }
        break;

        case 2: //enfermo con sintomas
            if (randomDeviceGenerator(semilla2, 0, 100) <= 90) {
                matrixProxima[i * columnas + j] = 1;
            } else {
                matrixProxima[i * columnas + j] = 2;
            }
            matrixProxima[i * columnas + j + 3] = matrix[i * columnas + j + 3] + 1;
            break;

        case 3: //enfermo sin sintomas
            if (matrix[i * columnas + j + 3] >= 6) {
                matrixProxima[i * columnas + j] = 2;
            } else {
                matrixProxima[i * columnas + j] = 3;
            }
            matrixProxima[i * columnas + j + 3] = matrix[i * columnas + j + 3] + 1;
        break;

        case 4: //sano
            if ((randomDeviceGenerator(semilla3, 0, 100) / 100) <= (porcentajeConSintomas(vecindarioDeMoore(matrix, filas, columnas, j, i)) + susceptibilidad(matrix[i * columnas + j + 1], matrix[i * columnas + j + 2])) * 0.6 + 0.05){
                matrixProxima[i * columnas + j] = 3;
            }
            matrixProxima[i * columnas + j + 3] = -1;
        break;
    }

	//herida aleatoria
    matrixProxima[i * columnas + j + 2] = randomDeviceGenerator(semilla4, 0, 1);

    /* -- FIN DE: CALCULAR EL PROXIMO ESTADO -- */
}

/*		-- KERNEL --		*/

__global__ void simular(
	short * matrix,			//matriz actual
	short * matrixProxima, 	//matriz proxima
	int dimension,			//dimension de la matriz cuadrada
	int cantBloques,		//cantidad de bloques
	int filasPorBloque,		//cantidad de filas que le corresponde calcular a cada bloque
	int moduloBloque,		//cantBloques % filasPorBloque
	int cantThreads,		//cantidad de threads en un bloque
	int columnasPorThread,	//cantidad de columnas que le corresponde calcular a cada thread
	int moduloThread,		//cantThreads % columnasPorThread

	//semillas para eventos random
	int semilla1, int semilla2, int semilla3, int semilla4
	){
    
	int bloqueId = blockIdx.x;
	int threadId = threadIdx.x;

    for(int i = bloqueId * filasPorBloque; i < (bloqueId + 1) * filasPorBloque; i++){
    	for(int j = threadId * columnasPorThread; j < (threadId + 1) * columnasPorThread; j++){
    		calcularEstado(
    			i,					//nro de fila
    			j,					//nro de columna
    			matrix,				//matriz actual
    			matrixProxima,		//matriz proxima
    			dimension,			//dimension de la matriz cuadrada

    			//semillas para eventos random
    			semilla1, semilla2, semilla3, semilla4
			);
    	}

		if(threadId < moduloThread){
			calcularEstado(
				i,											//nro de fila
				columnasPorThread*cantThreads + threadId,	//nro de columna
				matrix,										//matriz actual
				matrixProxima,								//matriz proxima
				dimension,									//dimension de la matriz cuadrada

    			//semillas para eventos random
				semilla1,semilla2, semilla3, semilla4
			);
		}
    }

    if(bloqueId < moduloBloque){
    	for(int j = threadId * columnasPorThread; j < (threadId + 1) * columnasPorThread; j++){
    		calcularEstado(
    			filasPorBloque * cantBloques + bloqueId,	//nro de fila
    			j,											//nro de columna
    			matrix,										//matriz actual
    			matrixProxima,								//matriz proxima
    			dimension,									//dimension de la matriz cuadrada

    			//semillas para eventos random
    			semilla1, semilla2, semilla3, semilla4
			);
    	}
    }

    if(bloqueId < moduloBloque && threadId < moduloThread){
		calcularEstado(
			filasPorBloque * cantBloques + bloqueId,	//nro de fila
			columnasPorThread*cantThreads + threadId,	//nro de columna
			matrix,										//matriz actual
			matrixProxima,								//matriz proxima
			dimension,									//dimension de la matriz cuadrada
	
			//semillas para eventos random
			semilla1, semilla2, semilla3, semilla4
		);
	}
}

/*		-- MAIN --		*/

int main(int argc, char * argv[]) {
    //necesario para generar numeros aleatorios
    srand(time(NULL));

	//argvs
    int filas = atoi(argv[1]);
    int semanas = atoi(argv[2]);
    int cantBloques = atoi(argv[3]);
    int cantThreads = atoi(argv[4]);

	//matriz inicial y su contraparte en gpu
    short * matrix = (short * ) malloc(filas * filas * 4 * sizeof(short));
    short * matrix_gpu;
    
    //matriz proxima y su contraparte en gpu
    short * matrixProxima = (short * ) malloc(filas * filas * 4 * sizeof(short));
    short * matrixProxima_gpu;

	//puntero auxiliar usado para intercambio
    short * auxiliar;

	//calculo de cuanto trabajo va a tener cada bloque
    int moduloBloque = filas % cantBloques;
    int filasPorBloque = floor(filas / cantBloques);

	//calculo de cuanto trabajo va a tener cada thread
    int moduloThread = filas % cantThreads;
    int columnasPorThread = floor(filas / cantThreads);

    //printf("Filas por bloque %d\n\t%d bloque/s tuvieron que calcular otra fila\n\nColumnas por thread %d\n\t%d thread/s tuvieron que calcular otra columna\n\n", filasPorBloque, moduloBloque, columnasPorThread, moduloThread);

    //arranca a medir el tiempo
    struct timespec begin, end;
    clock_gettime(CLOCK_REALTIME, & begin);

	//se inicializa el automata celular (en CPU)
    inicializar(matrix, filas, filas * 4);

	//se ubican las matrices en la memoria de la GPU
    gpuErrchk(hipMalloc((void ** ) & matrix_gpu, filas * filas * 4 * sizeof(short)));
    gpuErrchk(hipMalloc((void ** ) & matrixProxima_gpu, filas * filas * 4 * sizeof(short)));

    //CPU -> GPU 
    hipMemcpy(matrix_gpu, matrix, filas * filas * 4 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(matrixProxima_gpu, matrixProxima, filas * filas * 4 * sizeof(short), hipMemcpyHostToDevice);

    //simulacion!
    for (int generacion = 0; generacion < semanas; generacion++) {

        //ejecutar kernel
        simular <<<cantBloques,cantThreads>>>(
        	matrix_gpu,				//matriz actual
        	matrixProxima_gpu,		//matriz proxima
        	filas,					//dimension de la matriz
        	cantBloques,			//cantidad de bloques
        	filasPorBloque,			//cantidad de filas que le corresponde calcular a cada bloque
        	moduloBloque,			//cantBloques % filasPorBloque
        	cantThreads,			//cantidad de threads
        	columnasPorThread,		//cantidad de columnas que le corresponde calcular a cada thread 
        	moduloThread,			//cantThreads % columnasPorThread

        	//semillas para eventos random
        	randInRange(0, 9), randInRange(0, 9), randInRange(0, 9), randInRange(0, 9)
    	);


        //check si hubo error
        hipError_t err = hipGetLastError();
        if (hipGetLastError() != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(-1);
        }

		//intercambio de punteros
        auxiliar = matrix_gpu;
        matrix_gpu = matrixProxima_gpu;
        matrixProxima_gpu = auxiliar;
    }

	//finaliza de medir el tiempo tiempo
    clock_gettime(CLOCK_REALTIME, & end);
    long seconds = end.tv_sec - begin.tv_sec;
    long nanoseconds = end.tv_nsec - begin.tv_nsec;
    double elapsed = seconds + nanoseconds * 1e-9;
    printf("Tiempo medido: %.3f segundos.\n", elapsed);

	/*
	PARA HACERLO COMPARABLE A MPI

	como en mpi no se hace un gather al final
	se deja comentado los cudaMemcpyDeviceToHost
	*/
	
	//GPU -> CPU
	//gpuErrchk(cudaPeekAtLastError());
    //gpuErrchk(cudaMemcpy(matrixProxima, matrixProxima_gpu, filas * filas * 4 * sizeof(short), cudaMemcpyDeviceToHost));

    //gpuErrchk(cudaPeekAtLastError());
    //gpuErrchk(cudaMemcpy(matrix, matrix_gpu, filas * filas * 4 * sizeof(short), cudaMemcpyDeviceToHost));
	
	//print resultado final
    //printMatrix(matrix, filas, filas * 4);

    //frees
    free(matrix);
    free(matrixProxima);
    hipFree(matrix_gpu);
    hipFree(matrixProxima_gpu);
    return 0;
}